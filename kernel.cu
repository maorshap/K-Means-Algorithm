#include "hip/hip_runtime.h"
//
#include "KMeansHeader.h"


__global__ void pointsMovementCalKernel(int size, double* dev_initPointsCordinates,double* dev_pointsVelocityArr, double* dev_currentPointsCordinates, double time)
{
	int processId = threadIdx.x;
	dev_currentPointsCordinates[processId] = dev_initPointsCordinates[processId] + (dev_pointsVelocityArr[processId] * time);
}

boolean calPointsCordsCuda(double time, double* initPointsCordinates, double* pointsVelocityArr, double* currentPointsCordniates, int size)
{
	hipError_t cudaStatus;
	int counter = 0;

	cudaStatus = computePointsCordinates(time,initPointsCordinates, pointsVelocityArr, currentPointsCordniates, size);
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!"); fflush(stdout);
		return FALSE;
	}

	return TRUE;
}

void error(double* dev_currentPointsCordinates, double* dev_pointsVelocityArr, double* dev_initPointsCordinates)
{
	hipFree(dev_currentPointsCordinates);
	hipFree(dev_pointsVelocityArr);
	hipFree(dev_initPointsCordinates);
}

hipError_t computePointsCordinates(double time,double* initPointsCordinates , double* pointsVelocityArr, double* currentPointsCordniates, int size)
{
	hipError_t cudaStatus;
	double* dev_currentPointsCordinates = 0;
	double* dev_pointsVelocityArr = 0;
	double* dev_initPointsCordinates = 0;
	int parts = size / 1000;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); fflush(stdout);
		error(dev_currentPointsCordinates, dev_pointsVelocityArr, dev_initPointsCordinates);
	}
	
	// Allocate GPU buffers for Points vector    .
	cudaStatus = hipMalloc((void**)&dev_currentPointsCordinates, size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!"); fflush(stdout);
		error(dev_currentPointsCordinates, dev_pointsVelocityArr, dev_initPointsCordinates);
	}

	cudaStatus = hipMalloc((void**)&dev_pointsVelocityArr, size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!"); fflush(stdout);
		error(dev_currentPointsCordinates, dev_pointsVelocityArr, dev_initPointsCordinates);
	}

	cudaStatus = hipMalloc((void**)&dev_initPointsCordinates, size * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!"); fflush(stdout);
		error(dev_currentPointsCordinates, dev_pointsVelocityArr, dev_initPointsCordinates);
	}

	// Copy input vectors from host memory to GPU buffers.
	//cudaStatus = hipMemcpy(dev_middleResultArr, middleResultArr, RANGE_SIZE * NUM_OF_THREADS * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_currentPointsCordinates, currentPointsCordniates, size * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!"); fflush(stdout);
		error(dev_currentPointsCordinates, dev_pointsVelocityArr, dev_initPointsCordinates);
	}

	cudaStatus = hipMemcpy(dev_pointsVelocityArr, pointsVelocityArr, size * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!"); fflush(stdout);
		error(dev_currentPointsCordinates, dev_pointsVelocityArr, dev_initPointsCordinates);
	}

	cudaStatus = hipMemcpy(dev_initPointsCordinates, initPointsCordinates, size * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!"); fflush(stdout);
		error(dev_currentPointsCordinates, dev_pointsVelocityArr, dev_initPointsCordinates);
	}

	
	pointsMovementCalKernel << <parts, size/parts >> >(size, dev_initPointsCordinates, dev_pointsVelocityArr, dev_currentPointsCordinates, time);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); fflush(stdout);
		error(dev_currentPointsCordinates, dev_pointsVelocityArr, dev_initPointsCordinates);
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		error(dev_currentPointsCordinates, dev_pointsVelocityArr , dev_initPointsCordinates);
	}
	return cudaStatus;
}


